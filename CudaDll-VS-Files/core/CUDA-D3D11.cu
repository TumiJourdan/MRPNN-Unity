#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <string>

#include <hip/hip_runtime.h>
#include <d3d11.h>
#include <hip/hip_runtime.h>
#include <cuda_d3d11_interop.h>
#include "IUnityGraphics.h"
#include "IUnityInterface.h"
#include "IUnityGraphicsD3D11.h"
#include <hiprand/hiprand_kernel.h>

#include ""
#include "volume.hpp"

#include <thread>

using namespace std;
#define CUDA_CHECK(err) checkCudaError(err, __FILE__, __LINE__)

inline void checkCudaError(hipError_t err, const char* file, int line) {
    if (err != hipSuccess) {
        std::cerr << "CUDA error at " << file << ":" << line << " - "
            << hipGetErrorString(err) << std::endl;
        std::cerr << "Press Enter to exit..." << std::endl;
        std::cin.get(); // Wait for the user to press Enter
        exit(err); // Exit if there's an error
    }
}

class Logger {
public:
    Logger(const std::string& filename) {
        file.open(filename, std::ios::out | std::ios::app);  // Open in append mode
        if (!file) {
            std::cerr << "Failed to open log file: " << filename << std::endl;
        }
    }

    ~Logger() {
        if (file.is_open()) {
            file.close();
        }
    }

    void log(const std::string& message) {
        if (file.is_open()) {
            file << message << std::endl;
            file.flush();  // Force data to write to disk
        }
    }

    // Overload to log numeric values
    void log(float value) {
        if (file.is_open()) {
            file << value << std::endl;
            file.flush();  // Force data to write to disk
        }
    }
    void logArray(float* array, int width, int height) {
        if (file.is_open()) {
            for (int i = 0; i < height; ++i) {
                for (int j = 0; j < width; ++j) {
                    file << array[i * width + j]; // Access the element in a 1D manner
                    if (j < width - 1) {
                        file << ","; // Add a comma if not the last element in the row
                    }
                }
                file << std::endl; // New line after each row
            }
            file.flush(); // Force data to write to disk
        }
    }

private:
    std::ofstream file;
};

// Function pointer for Unity's texture update callback
static IUnityInterfaces* s_UnityInterfaces = nullptr;
static IUnityGraphics* s_Graphics = nullptr;
static UnityGfxRenderer s_RendererType = kUnityGfxRendererNull;
static ID3D11Device* g_D3D11Device = nullptr;
static IDXGIAdapter* g_DXGIAdapter = nullptr;
ID3D11DeviceContext* deviceContext = nullptr;
ID3D11Query* query;

// Function pointer for Unity's texture update callback
static void(UNITY_INTERFACE_API* UpdateTextureFromUnity)(void* textureHandle) = nullptr;

// CUDA graphics resource
vector<tuple<ID3D11Texture2D*, hipGraphicsResource*, hipSurfaceObject_t>> cudaResources;
hipGraphicsResource* infoResource;


// Logger (assuming you have a logger class)
Logger logger("Cuda_Unity_Direct.txt");
// Graphics device event callback
static void UNITY_INTERFACE_API OnGraphicsDeviceEvent(UnityGfxDeviceEventType eventType)
{
    switch (eventType)
    {
    case kUnityGfxDeviceEventInitialize:
    {
        s_RendererType = s_Graphics->GetRenderer();
        if (s_RendererType == kUnityGfxRendererD3D11)
        {
            g_D3D11Device = s_UnityInterfaces->Get<IUnityGraphicsD3D11>()->GetDevice();
            if (g_D3D11Device)
            {
                IDXGIDevice* dxgiDevice = nullptr;
                if (SUCCEEDED(g_D3D11Device->QueryInterface(__uuidof(IDXGIDevice), (void**)&dxgiDevice)))
                {
                    dxgiDevice->GetAdapter(&g_DXGIAdapter);
                    dxgiDevice->Release();
                }

                //Query setup
                D3D11_QUERY_DESC queryDesc;
                ZeroMemory(&queryDesc, sizeof(queryDesc));
                queryDesc.Query = D3D11_QUERY_EVENT;
                HRESULT result = g_D3D11Device->CreateQuery(&queryDesc, &query);
                if (FAILED(result)) {
                    logger.log("Failed to make query");
                }

                g_D3D11Device->GetImmediateContext(&deviceContext);

            }
            logger.log("D3D11 device and DXGI Adapter initialized");
        }
        break;
    }
    case kUnityGfxDeviceEventShutdown:
    {
        s_RendererType = kUnityGfxRendererNull;
        if (g_DXGIAdapter)
        {
            g_DXGIAdapter->Release();
            g_DXGIAdapter = nullptr;
        }
        g_D3D11Device = nullptr;
        break;
    }
    case kUnityGfxDeviceEventBeforeReset:
    case kUnityGfxDeviceEventAfterReset:
        // Handle these events if necessary
        break;
    };
}

// Plugin load event
extern "C" void UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API UnityPluginLoad(IUnityInterfaces * unityInterfaces)
{
    logger.log("Loading");
    s_UnityInterfaces = unityInterfaces;
    s_Graphics = s_UnityInterfaces->Get<IUnityGraphics>();
    s_Graphics->RegisterDeviceEventCallback(OnGraphicsDeviceEvent);

    // Run OnGraphicsDeviceEvent(initialize) manually on plugin load
    OnGraphicsDeviceEvent(kUnityGfxDeviceEventInitialize);
}
extern "C" void UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API deregister_cuda_resources() {
    for (auto& resourceTuple : cudaResources) {
        hipGraphicsResource* cudaResource = std::get<1>(resourceTuple);
        //destroy surf
        hipSurfaceObject_t surfObj = std::get<2>(resourceTuple);
        if (surfObj != 0) {
            hipError_t err = hipDestroySurfaceObject(surfObj);
            hipDeviceSynchronize();
            if (err != hipSuccess) {
                logger.log("Failed to destroy surface resource: 1 " + std::string(hipGetErrorString(err)));
            }
        }
        hipDeviceSynchronize();

        // Unmap the resource if it's mapped
        hipGraphicsUnmapResources(1, &cudaResource, 0);

        // Deregister the CUDA graphics resource
        hipError_t err = hipGraphicsUnregisterResource(cudaResource);
        if (err != hipSuccess) {
            logger.log("Failed to unregister resource: " + std::string(hipGetErrorString(err)));
        }
    }

    // Clear the resources vector
    cudaResources.clear();
}
// Plugin unload event
extern "C" void UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API UnityPluginUnload()
{
    logger.log("Starting plugin unload");

    // Do full cleanup including unregistering resources
    deregister_cuda_resources();

    if (s_Graphics != nullptr) {
        s_Graphics->UnregisterDeviceEventCallback(OnGraphicsDeviceEvent);
        logger.log("Graphics device callback unregistered");
    }

    logger.log("Plugin unload completed");
}

// Set texture update callback
extern "C" void UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API SetTextureFromUnity(void* textureHandle)
{
    UpdateTextureFromUnity = (void(UNITY_INTERFACE_API*)(void*))textureHandle;
    logger.log("Stored function pointer for texture update");
}
// Initialize CUDA interop

extern "C" void UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API RegisterTexture2D(ID3D11Texture2D * texture) {
    if (!g_DXGIAdapter) {
        logger.log("DXGI Adapter not initialized");
        return;
    }
    // Get the CUDA device corresponding to the D3D11 device
    int cudaDevice;
    hipError_t err = cudaD3D11GetDevice(&cudaDevice, g_DXGIAdapter);
    if (err != hipSuccess) {
        logger.log("Failed to get CUDA device: " + std::string(hipGetErrorString(err)));
        return;
    }

    // Set the CUDA device
    err = hipSetDevice(cudaDevice);
    if (err != hipSuccess) {
        logger.log("Failed to set CUDA device: " + std::string(hipGetErrorString(err)));
        return;
    }

    // Check if the texture is already registered
    for (const auto& resourceTuple : cudaResources) {
        if (std::get<0>(resourceTuple) == texture) {
            logger.log("Texture is already registered with CUDA");
            return;
        }
    }

    // Register the D3D11 texture with CUDA and add to the vector
    hipGraphicsResource* cudaResource = nullptr;
    err = cudaGraphicsD3D11RegisterResource(&cudaResource, texture, hipGraphicsRegisterFlagsNone);
    if (err != hipSuccess) {
        logger.log("Failed to register texture with CUDA: " + std::string(hipGetErrorString(err)));
        return;
    }

    // Store the newly registered resource in the vector
    cudaResources.emplace_back(texture, cudaResource, 0);
    logger.log("CUDA interop initialized successfully and resource added to vector");
}

extern "C" void UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API unmap_cuda_resources() {

    int count = 0;
    for (auto& resourceTuple : cudaResources) {
        hipGraphicsResource* cudaResource = std::get<1>(resourceTuple);
        // Unmap the CUDA graphics resource
        hipError_t err1 = hipGraphicsUnmapResources(1, &std::get<1>(resourceTuple), 0);

        if (err1 != hipSuccess) {
            logger.log("Failed to unmap resource: 2 " + std::string(hipGetErrorString(err1)));
            logger.log("Count = " + to_string(count));
        }
        count += 1;

    }

    hipDeviceSynchronize();
}


// GPU Memory buffers
float3* d_target;
Histogram* d_histo_buffer;
unsigned int* d_target2;

// Volume instance
VolumeRender* volume_inst = nullptr;
int multiScatter = 512;
float g = 0.857;
int randseed = 0;
VolumeRender::RenderType rt = VolumeRender::MRPNN;
int toneType = 2;//ACES tone
bool denoise = true;

//Debug histo
__device__ __managed__ float3 prevOrigin;
__device__ __managed__ float3 prevUp;
__device__ __managed__ float3 prevRight;
__device__ __managed__ float3 prevLight;
__device__ __managed__ float prevAlpha;
__device__ __managed__ float3 prevColour;
extern "C" void UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API initialize_memory(int2 size,int resolution = 512) {


    hipDeviceSynchronize();
    volume_inst = new VolumeRender(resolution);

    CUDA_CHECK(hipMalloc((void**)&d_target, size.x * size.y * sizeof(float3)));
    CUDA_CHECK(hipMalloc((void**)&d_histo_buffer, size.x * size.y * sizeof(Histogram)));
    CUDA_CHECK(hipMalloc((void**)&d_target2, size.x * size.y * sizeof(unsigned int)));
    prevOrigin = make_float3(0.0f, 0.0f, 0.0f);
    prevUp = make_float3(0.0f, 0.0f, 0.0f);
    prevRight = make_float3(0.0f, 0.0f, 0.0f);
    hipDeviceSynchronize();
    volume_inst->SetDatas([](int x, int y, int z, float u, float v, float w) {
        float dis = distance(make_float3(0.5f, 0.5f, 0.5f), make_float3(u, v, w));
        return dis < 0.25 ? 1.0f : 0;
        });

    volume_inst->Update(); // Call Update after changing volumetric data.

}

extern "C" void UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API initialize_memory_path(int2 size, const char* path, int targetResolution) {
    
    try {
        std::string pathStr(path);
        // Replace all backslashes with forward slashes
        std::replace(pathStr.begin(), pathStr.end(), '\\', '/');

        std::string fullPath = pathStr + ".bin";
        logger.log("Attempting to load model from: " + fullPath);

        FILE* file = fopen(fullPath.c_str(), "rb");
        if (file == nullptr) {
            logger.log("ERROR: Cannot open file at path: " + fullPath);
            logger.log("Error code: " + std::string(strerror(errno)));
            return;
        }
        fclose(file);

        volume_inst = new VolumeRender(pathStr, targetResolution);
    }
    catch (const std::exception& e) {
        logger.log("Exception: " + std::string(e.what()));
    }

    CUDA_CHECK(hipMalloc((void**)&d_target, size.x * size.y * sizeof(float3)));
    CUDA_CHECK(hipMalloc((void**)&d_histo_buffer, size.x * size.y * sizeof(Histogram)));
    CUDA_CHECK(hipMalloc((void**)&d_target2, size.x * size.y * sizeof(unsigned int)));
    prevOrigin = make_float3(0.0f, 0.0f, 0.0f);
    prevUp = make_float3(0.0f, 0.0f, 0.0f);
    prevRight = make_float3(0.0f, 0.0f, 0.0f);
    hipDeviceSynchronize();


}
extern "C" void UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API set_volume_datas() {


    float3 scatter = float3{ 1, 1, 1 };
    volume_inst->SetScatterRate(scatter);
    volume_inst->UpdateHGLut(0.857);
    volume_inst->SetEnvExp(0);
    volume_inst->SetTrScale(1);
    volume_inst->SetExposure(1);
    volume_inst->SetSurfaceIOR(-1);
    volume_inst->SetCheckboard(true);
}

extern "C" void UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API set_raymarch_step(int stepCount) {
    volume_inst->set_Step_Num(stepCount);
}
extern "C" void UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API clean_memory() {
    Logger logger("cleanup.txt");
    logger.log("Starting play mode cleanup");

    hipError_t err = hipSuccess;
    volume_inst->saveTimings();
    // Clean up volume instance if it exists
    if (volume_inst != nullptr) {
        delete volume_inst;
        volume_inst = nullptr;
        logger.log("Volume instance cleaned up");
    }

    // Free device memory
    if (d_target != nullptr) {
        err = hipFree(d_target);
        if (err != hipSuccess) {
            logger.log("Warning: Failed to free d_target: " + std::string(hipGetErrorString(err)));
        }
        d_target = nullptr;
    }

    if (d_target2 != nullptr) {
        err = hipFree(d_target2);
        if (err != hipSuccess) {
            logger.log("Warning: Failed to free d_target2: " + std::string(hipGetErrorString(err)));
        }
        d_target2 = nullptr;
    }

    if (d_histo_buffer != nullptr) {
        err = hipFree(d_histo_buffer);
        if (err != hipSuccess) {
            logger.log("Warning: Failed to free d_histo_buffer: " + std::string(hipGetErrorString(err)));
        }
        d_histo_buffer = nullptr;
    }

    // Reset CUDA graphics resource but don't unregister it
    // since we might need it again in the next play session
    unmap_cuda_resources();
    deregister_cuda_resources();
    hipDeviceSynchronize();
    logger.log("Play mode cleanup completed");
}

extern "C" void UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API setHDRIInterface(string path) {
    volume_inst->SetHDRI(path);
}

__host__ __device__ bool hasChanged(float3 a, float3 b) {
    return a.x != b.x || a.y != b.y || a.z != b.z;
}

extern "C" void UNITY_INTERFACE_EXPORT UNITY_INTERFACE_API cuda_run_render(float3 origin, float3 up, float3 right, float3 lightDir, float3 lightColor, float alpha, int2 size) {
    //DIRCT3D INTEROP


    hipError_t err = hipSuccess;
    hipDeviceSynchronize();
    
    for (auto& resourceTuple : cudaResources) {
        hipGraphicsResource* cudaResource = std::get<1>(resourceTuple);

        // Map the resource
        err = hipGraphicsMapResources(1, &cudaResource, 0);
        if (err != hipSuccess) {
            logger.log("Failed to map resource: " + std::string(hipGetErrorString(err)));
            continue;
        }
        hipDeviceSynchronize();
        // Create the surface object if needed
        hipSurfaceObject_t& surfObj = std::get<2>(resourceTuple);
        if (surfObj == 0) {
            // Get the CUDA array
            hipArray* cuArray = nullptr;
            err = hipGraphicsSubResourceGetMappedArray(&cuArray, cudaResource, 0, 0);
            if (err != hipSuccess) {
                logger.log("Failed to get mapped array: " + std::string(hipGetErrorString(err)));
                hipGraphicsUnmapResources(1, &cudaResource, 0);
                continue;
            }
            hipDeviceSynchronize();
            hipResourceDesc resDesc = {};
            resDesc.resType = hipResourceTypeArray;
            resDesc.res.array.array = cuArray;

            err = hipCreateSurfaceObject(&surfObj, &resDesc);
            if (err != hipSuccess) {
                logger.log("Failed to create surface object: " + std::string(hipGetErrorString(err)));
                hipGraphicsUnmapResources(1, &cudaResource, 0);
                continue;
            }
            hipDeviceSynchronize();
        }
    }
    //CALLING THE RENDER FUNCTION


    randseed += 1;
    //reset randseed 
    if (hasChanged(origin, prevOrigin) || hasChanged(up, prevUp) || hasChanged(right, prevRight) || hasChanged(lightDir, prevLight) || hasChanged(prevColour, lightColor) || prevAlpha!=alpha) {
        randseed = 0;
    }
    // Update the previous frame values
    prevOrigin = origin;
    prevUp = up;
    prevRight = right;
    prevLight = lightDir;
    prevAlpha = alpha;
    prevColour = lightColor;
    hipSurfaceObject_t surfObj = get<2>(cudaResources[0]);
    hipSurfaceObject_t infoSurfObj = get<2>(cudaResources[1]);

    volume_inst->Render(d_target, d_histo_buffer, d_target2, size, origin, up, right, lightDir, lightColor, alpha, multiScatter, 0.857, randseed, rt, toneType, denoise, surfObj,infoSurfObj);

    //CLEANING DIRECT3D INTEROP
    unmap_cuda_resources();
    CUDA_CHECK(hipDeviceSynchronize());  // Wait for kernel to complete
}


/*

RESOURCES

unsigned int* h_target2 = nullptr;

// GPU Memory buffers
float3* d_target;
Histogram* d_histo_buffer;
unsigned int* d_target2;
VolumeRender* volume_inst = nullptr;
mapped pointer
registered resource

*/